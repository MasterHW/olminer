#include "hip/hip_runtime.h"
#include "olhash_cuda_miner_kernel.h"

#include "olhash_cuda_miner_kernel_globals.h"

#include "cuda_helper.h"

#define copy(dst, src, count)        \
    for (int i = 0; i != count; ++i) \
    {                                \
        (dst)[i] = (src)[i];         \
    }

#include "olhash.cuh"

__global__ void olhash_search(volatile Search_results* g_output, uint64_t start_nonce)
{
    uint32_t const gid = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t distance;
    if (compute_distance(start_nonce + gid, &distance))
        return;
    uint32_t index = atomicInc((uint32_t*)&g_output->count, 0xffffffff);
    if (index >= MAX_SEARCH_RESULTS)
        return;
    g_output->result[index].gid = gid;
    g_output->result[index].distance = distance;
}

void run_olhash_search(uint32_t gridSize, uint32_t blockSize, hipStream_t stream,
                       volatile Search_results* g_output, uint64_t start_nonce)
{
    olhash_search<<<gridSize, blockSize, 0, stream>>>(g_output, start_nonce);
    CUDA_SAFE_CALL(hipGetLastError());
}

void set_common_data(const std::vector<uint8_t>& _work,
                     const std::vector<uint8_t>& _miner_key,
                     const std::vector<uint8_t>& _merkle_root,
                     const std::vector<uint8_t>& _timestamp)
{
  const uint64_t miner_key_size = _miner_key.size();
  const uint64_t timestamp_size = _timestamp.size();
  const uint64_t nonce_offset = miner_key_size + 64;
  const uint64_t hash_template_length = nonce_offset + 64 + timestamp_size;

  unsigned char hash_template[4*64];
  memset(hash_template, 0, 4*64);
  memcpy(hash_template, _miner_key.data(), miner_key_size);
  memcpy(hash_template + miner_key_size, _merkle_root.data(), 64);
  memcpy(hash_template + miner_key_size + 2*64, _timestamp.data(), timestamp_size);
  
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_work), _work.data(), 64));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_hash_template), hash_template, 4*64));

  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_nonce_offset), &nonce_offset, sizeof(uint64_t)));
  CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_hash_template_length), &hash_template_length, sizeof(uint64_t)));
}

void set_target(uint64_t _target)
{
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_target), &_target, sizeof(uint64_t)));
}
